/*
Run this file

nvcc single_test.cu -o test.app \
    -I/nix/store/ca1xhh6rxcx0a23qnsla0gsg44vwgwl4-cuda-merged-12.2/include \
    -L/nix/store/ca1xhh6rxcx0a23qnsla0gsg44vwgwl4-cuda-merged-12.2/lib \
    -lcudart && ./test.app
*/
#include <hip/hip_runtime.h>

#include <iostream>

int main() {
  int run_version, driver_version;
  std::cout << "Return Code Runtime Version: " << hipRuntimeGetVersion(&run_version) << "\n";
  std::cout << "Return Code Driver Version: " << hipDriverGetVersion(&driver_version) << "\n";
  std::cout << "Runtime Version: " << run_version << "\n";
  std::cout << "Driver Version: " << driver_version << "\n";

  return 0;
}
