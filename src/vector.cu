#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <iostream>

#include "vector.hpp"

// CUDA kernel for vector addition
__global__ void vectorAddKernel(const float* A, const float* B, float* C, int N) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < N) {
    C[i] = A[i] + B[i];
  }
}

// Vector addition function
void vectorAdd(const float* h_A, const float* h_B, float* h_C, int N) {
  size_t size = N * sizeof(float);

  // Allocate device memory
  float* d_A;
  float* d_B;
  float* d_C;
  hipMalloc((void**)&d_A, size);
  hipMalloc((void**)&d_B, size);
  hipMalloc((void**)&d_C, size);

  // Copy data from host to device
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

  // Launch the kernel
  int threadsPerBlock = 256;
  int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
  vectorAddKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    std::cerr << "CUDA kernel launch error: " << hipGetErrorString(err) << std::endl;
  }
  // Synchronize device
  hipDeviceSynchronize();

  // Copy result back to host
  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

  // Free device memory
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}
