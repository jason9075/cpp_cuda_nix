#include <hip/hip_runtime.h>

#include <iostream>

#include "version.hpp"

void printVersion() {
  int run_version, driver_version;
  std::cout << "Return Code Runtime Version: " << hipRuntimeGetVersion(&run_version) << "\n";
  std::cout << "Return Code Driver Version: " << hipDriverGetVersion(&driver_version) << "\n";
  std::cout << "Runtime Version: " << run_version << "\n";
  std::cout << "Driver Version: " << driver_version << "\n";

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    std::cerr << "CUDA kernel launch error: " << hipGetErrorString(err) << std::endl;
  }
}
